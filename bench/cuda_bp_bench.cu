#include "hip/hip_runtime.h"
/**
 * Copyright 2018-present Antonio Mallia <me@antoniomallia.it>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gmock/gmock.h>
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include "bp/utils.hpp"

#include "synthetic/uniform.hpp"
#include "benchmark/benchmark.h"

#include "bp/cuda_bp.cuh"
#include "bp/cuda_common.hpp"

__global__
void warmUpGPU()
{
  // do nothing
}

class RandomValuesFixture : public ::benchmark::Fixture {

    static std::vector<uint32_t> generate_random_vector(size_t n) {
        std::random_device rd;
        std::mt19937 gen(rd());
        std::vector<uint32_t> values(n);
        std::uniform_int_distribution<> dis(uint32_t(0));
        std::generate(values.begin(), values.end(), [&](){ return dis(gen); });
        return values;
    }

public:
    using ::benchmark::Fixture::SetUp;
    using ::benchmark::Fixture::TearDown;

    virtual void SetUp(::benchmark::State& st) {
        values = generate_random_vector(st.range(0));
        std::sort(values.begin(), values.end());
        utils::delta_encode(values.data(), values.size());

        encoded_values.resize(values.size() * 8);
        auto compressedsize = cuda_bp::encode(encoded_values.data(), values.data(), values.size());
        encoded_values.resize(4*compressedsize);
        encoded_values.shrink_to_fit();

        decoded_values.resize(values.size());
        CUDA_CHECK_ERROR(hipSetDevice(3));
        warmUpGPU<<<1, 1>>>();
        CUDA_CHECK_ERROR(hipMalloc((void **)&d_encoded, encoded_values.size() * sizeof(uint8_t)));
        CUDA_CHECK_ERROR(hipMemcpy(d_encoded, encoded_values.data(), encoded_values.size() * sizeof(uint8_t), hipMemcpyHostToDevice));

        CUDA_CHECK_ERROR(hipMalloc((void **)&d_decoded, values.size() * sizeof(uint32_t)));

    }

    virtual void TearDown(::benchmark::State&) {
        CUDA_CHECK_ERROR(hipMemcpy(decoded_values.data(), d_decoded, values.size() * sizeof(uint32_t), hipMemcpyDeviceToHost));

        ASSERT_EQ(decoded_values.size(), values.size());
        for (size_t i = 0; i < values.size(); ++i)
        {
            ASSERT_EQ(decoded_values[i], values[i]);
        }
        hipFree(d_encoded);
        hipFree(d_decoded);
        values.clear();
        encoded_values.clear();
        decoded_values.clear();
    }
    std::vector<uint32_t> values;
    std::vector<uint8_t> encoded_values;
    std::vector<uint32_t> decoded_values;
    uint8_t *  d_encoded;
    uint32_t * d_decoded;
};


BENCHMARK_DEFINE_F(RandomValuesFixture, decode)(benchmark::State& state) {
    while (state.KeepRunning()) {
        cuda_bp::decode(d_decoded, d_encoded, decoded_values.size());
    }
    auto bpi = double(encoded_values.size())/decoded_values.size();
    state.counters["bpi"] = benchmark::Counter(bpi, benchmark::Counter::kAvgThreads);
}
BENCHMARK_REGISTER_F(RandomValuesFixture, decode)->Range(1ULL<<14, 1ULL<<28);

BENCHMARK_MAIN();
