#include "hip/hip_runtime.h"
/**
 * Copyright 2018-present Antonio Mallia <me@antoniomallia.it>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gmock/gmock.h"
#include "gtest/gtest.h"
#include "benchmark/benchmark.h"

#include "bp/cuda_copy.cuh"
#include "bp/utils/utils.hpp"
#include "bp/utils/cuda_utils.hpp"

#include <hip/hip_runtime.h>

__global__
void warmUpGPU()
{
  // do nothing
}

class RandomValuesFixture : public ::benchmark::Fixture {

    static std::vector<uint32_t> generate_random_vector(size_t n) {
        std::random_device rd;
        std::mt19937 gen(rd());
        std::vector<uint32_t> values(n);
        std::uniform_int_distribution<> dis(uint32_t(0));
        std::generate(values.begin(), values.end(), [&](){ return dis(gen); });
        return values;
    }

public:
    using ::benchmark::Fixture::SetUp;
    using ::benchmark::Fixture::TearDown;

    virtual void SetUp(::benchmark::State& st) {
        values = generate_random_vector(st.range(0));

        decoded_values.resize(values.size());
        CUDA_CHECK_ERROR(hipSetDevice(0));
        warmUpGPU<<<1, 1>>>();
        CUDA_CHECK_ERROR(hipMalloc((void **)&d_encoded, values.size() * sizeof(uint32_t)));
        CUDA_CHECK_ERROR(hipMemcpy(d_encoded, values.data(), values.size() * sizeof(uint32_t), hipMemcpyHostToDevice));

        CUDA_CHECK_ERROR(hipMalloc((void **)&d_decoded, values.size() * sizeof(uint32_t)));
    }

    virtual void TearDown(::benchmark::State&) {
        CUDA_CHECK_ERROR(hipMemcpy(decoded_values.data(), d_decoded, values.size() * sizeof(uint32_t), hipMemcpyDeviceToHost));

        ASSERT_EQ(decoded_values.size(), values.size());
        for (size_t i = 0; i < values.size(); ++i)
        {
            ASSERT_EQ(decoded_values[i], values[i]);
        }
        CUDA_CHECK_ERROR(hipFree(d_encoded));
        CUDA_CHECK_ERROR(hipFree(d_decoded));
        values.clear();
        decoded_values.clear();
    }
    std::vector<uint32_t> values;
    std::vector<uint32_t> decoded_values;
    uint8_t *  d_encoded;
    uint32_t * d_decoded;
};


BENCHMARK_DEFINE_F(RandomValuesFixture, decode)(benchmark::State& state) {
    while (state.KeepRunning()) {
        cuda_copy::decode(d_decoded, reinterpret_cast<uint8_t*>(d_encoded), decoded_values.size());
    }
    auto bpi = 32;
    state.counters["bpi"] = benchmark::Counter(bpi, benchmark::Counter::kAvgThreads);
}
BENCHMARK_REGISTER_F(RandomValuesFixture, decode)->Range(1ULL<<14, 1ULL<<28);

BENCHMARK_MAIN();
